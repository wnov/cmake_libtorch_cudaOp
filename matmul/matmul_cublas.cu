#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "matmul.h"

void matmul_cublas(float *A, float *B, float *C, int m, int n, int k,
                   const float alpha, const float beta) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, B, n, A, k,
              &beta, C, n);
}
